#include "hip/hip_runtime.h"
/*
Copyright (c) 2011-2013 Gerhard Reitmayr, TU Graz

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the
"Software"), to deal in the Software without restriction, including
without limitation the rights to use, copy, modify, merge, publish,
distribute, sublicense, and/or sell copies of the Software, and to
permit persons to whom the Software is furnished to do so, subject to
the following conditions:

The above copyright notice and this permission notice shall be included
in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include "kfusion.h"
#include "perfstats.h"
#include <iostream>

using namespace std;

PerfStats Stats;

__global__ void setSphere( Volume volume, const float3 center, const float radius, const float val ){
    uint3 pos = make_uint3(thr2pos2());
    for(pos.z = 0; pos.z < volume.size.z; ++pos.z) {
        const float d = length(volume.pos(pos) - center);
        if(d < radius)
            volume.set(pos, make_float2(val, 0.0f));
    }
}

__global__ void setBox( Volume volume, const float3 min_corner, const float3 max_corner, const float val ){
    uint3 pos = make_uint3(thr2pos2());
    for(pos.z = 0; pos.z < volume.size.z; ++pos.z) {
        const float3 p = volume.pos(pos);
        if(min_corner.x < p.x && min_corner.y < p.y && min_corner.z < p.z && 
           p.x < max_corner.x && p.y < max_corner.y && p.z < max_corner.z )
            volume.set(pos, make_float2(val, 0.0f));
    }
}

void initVolumeWrap( Volume volume, const float val ){
    dim3 block(32,16);
    initVolume<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, make_float2(val, 0.0f));
}

void setBoxWrap(Volume volume, const float3 min_corner, const float3 max_corner, const float val ){
    dim3 block(32,16);
    setBox<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, min_corner, max_corner, val);
}

void setSphereWrap(Volume volume, const float3 center, const float radius, const float val ){
    dim3 block(32,16);
    setSphere<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, center, radius, val);
}

__global__ void renderNormals( Image<uchar3> out, const Image<float3> in ){
    float3 n = in.el();
    if(n.x == -2)
        out.el() = make_uchar3(0,0,0);
    else {
        n = normalize(n);
        out.el() = make_uchar3(n.x*128 + 128, n.y*128+128, n.z*128+128);
    }
}

void renderNormalMap( Image<uchar3> out, const Image<float3> & normal ){
    dim3 block(20,20);
    renderNormals<<<divup(normal.size, block), block>>>( out, normal );
}

__global__ void renderLightKernel( Image<uchar4> out, const Image<float3> vertex, const Image<float3> normal, const float3 light, const float3 ambient ){
    if(normal.el().x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 diff = normalize(light - vertex.el());
        const float dir = fmaxf(dot(normal.el(), diff), 0.f);
        const float3 col = clamp(make_float3(dir) + ambient, 0.f, 1.f) * 255;
        out.el() = make_uchar4(col.x, col.y, col.z, 255);
    }
}

void renderLight( Image<uchar4> out, const Image<float3> & vertex, const Image<float3> & normal, const float3 light, const float3 ambient ){
    dim3 block(32,16);
    renderLightKernel<<<divup(normal.size, block), block>>>( out, vertex, normal, light, ambient );
}

__global__ void renderTextureKernel( Image<uchar4> out, const Image<float3> vertex, const Image<float3> normal, const Image<uchar3> texture, const Matrix4 texproj, const float3 light){
    if(normal.el().x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 proj = texproj * vertex.el();
        const float2 projPixel = make_float2( proj.x / proj.z + 0.5f, proj.y / proj.z + 0.5f);
        
        const float3 diff = normalize(light - vertex.el());
        const float dir = fmaxf(dot(normal.el(), diff), 0.f); // * 255;
        if(projPixel.x < 0 || projPixel.x > texture.size.x-1 || projPixel.y < 0 || projPixel.y > texture.size.y-1 ){
            out.el() = make_uchar4(dir*255,dir*255,dir*255,255);
        } else {
            const uchar3 texcol = texture[make_uint2(projPixel.x, projPixel.y)];
            out.el() = make_uchar4(texcol.x*dir, texcol.y*dir, texcol.z*dir, 255);
        }
    }
}

void renderTexture( Image<uchar4> out, const Image<float3> & vertex, const Image<float3> & normal, const Image<uchar3> & texture, const Matrix4 & texproj, const float3 light){
    dim3 block(32,16);
    renderTextureKernel<<<divup(normal.size, block), block>>>( out, vertex, normal, texture, texproj, light);
}

__global__ void renderDepth( Image<uchar3> out, const Image<float> depth, const float nearPlane, const float farPlane){
    const float d = (clamp(depth.el(), nearPlane, farPlane) - nearPlane) / (farPlane - nearPlane);
    out.el() = make_uchar3(d * 255, d * 255, d * 255);
}

void renderDepthMap( Image<uchar3> out, const Image<float> & depth, const float nearPlane, const float farPlane ){
    dim3 block(32,16);
    renderDepth<<<divup(depth.size, block), block>>>( out, depth, nearPlane, farPlane );
}

__global__ void renderTrack( Image<uchar4> out, const Image<TrackData> data ){
    const uint2 pos = thr2pos2();
    switch(data[pos].result){
    case 1: out[pos] = make_uchar4(128, 128, 128,0);  // ok
        break;
    case -1: out[pos] = make_uchar4(0, 0, 0,0);      // no input
        break;
    case -2: out[pos] = make_uchar4(255,0,0,0);        // not in image
        break;
    case -3:  out[pos] = make_uchar4(0,255,0,0);        // no correspondence
        break;
    case -4: out[pos] = make_uchar4(0,0,255,0);        // to far away
        break;
    case -5: out[pos] = make_uchar4(255,255,0,0);     // wrong normal
        break;
    }
}

void renderTrackResult( Image<uchar4> out, const Image<TrackData> & data ){
    dim3 block(32,16);
    renderTrack<<<divup(out.size, block), block>>>( out, data );
}

__global__ void raycastLight( Image<uchar4> render, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const float3 ambient){
    const uint2 pos = thr2pos2();
    
    float4 hit = raycast( volume, pos, view, nearPlane, farPlane, step, largestep);
    if(hit.w > 0){
        const float3 test = make_float3(hit);
        const float3 surfNorm = volume.grad(test);
        if(length(surfNorm) > 0){
            const float3 diff = normalize(light - test);
            const float dir = fmaxf(dot(normalize(surfNorm), diff), 0.f);
            const float3 col = clamp(make_float3(dir) + ambient, 0.f, 1.f) * 255;
            render.el() = make_uchar4(col.x, col.y, col.z,0);
        } else {
            render.el() = make_uchar4(0,0,0,0);
        }
    } else {
        render.el() = make_uchar4(0,0,0,0);
    }
}

void renderVolumeLight( Image<uchar4> out, const Volume & volume, const Matrix4 view, const float nearPlane, const float farPlane, const float largestep, const float3 light, const float3 ambient ){
    dim3 block(16,16);
    raycastLight<<<divup(out.size, block), block>>>( out,  volume, view, nearPlane, farPlane, volume.dim.x/volume.size.x, largestep, light, ambient );
}

__global__ void raycastInput( Image<float3> pos3D, Image<float3> normal, Image<float> depth, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep){
    const uint2 pos = thr2pos2();
    
    float4 hit = raycast( volume, pos, view, nearPlane, farPlane, step, largestep);
    if(hit.w > 0){
        pos3D[pos] = make_float3(hit);
        depth[pos] = hit.w;
        float3 surfNorm = volume.grad(make_float3(hit));
        if(length(surfNorm) == 0){
            normal[pos].x = -2;
        } else {
            normal[pos] = normalize(surfNorm);
        }
    } else {
        pos3D[pos] = make_float3(0);
        normal[pos] = make_float3(0);
        depth[pos] = 0;
    }
}

void renderInput( Image<float3> pos3D, Image<float3> normal, Image<float> depth, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep){
    dim3 block(16,16);
    raycastInput<<<divup(pos3D.size, block), block>>>(pos3D, normal, depth, volume, view, nearPlane, farPlane, step, largestep);
}

__global__ void renderFusedKernel(Image<uint16_t> out, const Image<float3> vertex, const Matrix4 invPose) {
	const uint2 pos = thr2pos2();
//	float3 v = invPose * vertex[pos];
	float3 v = vertex[pos];
	float f = v.z;
	out[pos] = uint16_t (f * 1000);
}

void renderFusedMap(Image<uint16_t> out, const Image<float3> & vertex, const Matrix4 &invPose) {
	dim3 block(16,16);
	renderFusedKernel<<<divup(out.size, block), block>>>(out, vertex, invPose);
	hipDeviceSynchronize();
}
