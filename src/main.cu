#include "hip/hip_runtime.h"
/*
Copyright (c) 2011-2013 Gerhard Reitmayr, TU Graz

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the
"Software"), to deal in the Software without restriction, including
without limitation the rights to use, copy, modify, merge, publish,
distribute, sublicense, and/or sell copies of the Software, and to
permit persons to whom the Software is furnished to do so, subject to
the following conditions:

The above copyright notice and this permission notice shall be included
in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include "kfusion.h"
#include "helpers.h"

#include <iostream>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <cstring>
#include <dirent.h>
#include <cerrno>
#include <cmath>

#include <png++/png.hpp>
#include <jpeglib.h>

using namespace std;
using namespace TooN;

KFusion kfusion;
Image<uint16_t, HostDevice> depthImage;

SE3<float> initPose;
Matrix4 second_pose;

float size;
bool stop_run = false;

/*============================================================================*/

Image<uint16_t, HostDevice> fusedDepth;

////////////////////////////////////////////////////////////////////////////////
// global parameter

int   param_start_index = -1;

int   param_volume_size = 640;
float param_volume_dimension = 4.f;

int   param_frame_threshold = 11;
float param_angle_factor = 1.f;
float param_translation_factor = 1.f;
float param_rsme_threshold = 1.5e-2f;

int   param_file_name_length = 24;
int   param_time_stamp_pose = 8;
int   param_time_stamp_length = 12;

enum  KinfuMode {KINFU_FORWARD, KINFU_BACKWARD};
KinfuMode param_mode = KINFU_FORWARD;

// voxel resolution: 0.01 meter

////////////////////////////////////////////////////////////////////////////////

int file_index;
float angle_threshold, translation_threshold;

const int kImageRows = 480;
const int kImageCols = 640;
const int kImageChannels = 3;

vector<string> image_list;
vector<string> depth_list;
vector<string> extrinsic_list;

#ifdef INITIAL_POSE
vector<Matrix4> extrinsic_poses;
#endif

string data_dir, image_dir, depth_dir, fused_dir, extrinsic_dir;

////////////////////////////////////////////////////////////////////////////////

void GetFileNames(const string dir, vector<string> *file_list) {
  DIR *dp;
  struct dirent *dirp;
  if((dp  = opendir(dir.c_str())) == NULL) {
      cout << "Error(" << errno << ") opening " << dir << endl;
  }

  while ((dirp = readdir(dp)) != NULL) {
      file_list->push_back(dir + string(dirp->d_name));
  }
  closedir(dp);

  sort( file_list->begin(), file_list->end() );
  file_list->erase(file_list->begin()); //.
  file_list->erase(file_list->begin()); //..
}

////////////////////////////////////////////////////////////////////////////////

bool GetDepthData(string file_name, uint16_t *data) {
  png::image< png::gray_pixel_16 > img(file_name.c_str(),
      png::require_color_space< png::gray_pixel_16 >());

  int index = 0;
  for (int i = 0; i < kImageRows; ++i) {
    for (int j = 0; j < kImageCols; ++j) {
      uint16_t s = img.get_pixel(j, i);
      *(data + index) = (s << 13 | s >> 3);
      ++index;
    }
  }

  return true;
}

////////////////////////////////////////////////////////////////////////////////

void SaveFusedDepthFile() {
	string depth_full_name = depth_list[param_start_index];
	string depth_serial_name = depth_full_name.substr(
			depth_full_name.size() - param_file_name_length, param_file_name_length);
	string fused_full_name = fused_dir + depth_serial_name;

#ifdef RESOLUTION_1280X960
	png::image<png::gray_pixel_16> img(kImageCols * 2, kImageRows * 2);

	kfusion.Raycast_2();
	renderFusedMap(fusedDepth.getDeviceImage(), kfusion.vertex_2);

	for (int i = 0; i < kImageRows * 2; ++i) {
		for (int j = 0; j < kImageCols * 2; ++j) {
			uint16_t s = fusedDepth[make_uint2(j,i)];
			img[i][j] = (s >> 13 | s << 3);
		}
	}
#else
	png::image<png::gray_pixel_16> img(kImageCols, kImageRows);

	renderFusedMap(fusedDepth.getDeviceImage(), kfusion.vertex);
	hipDeviceSynchronize();

	for (int i = 0; i < kImageRows; ++i) {
		for (int j = 0; j < kImageCols; ++j) {
			uint16_t s = fusedDepth[make_uint2(j,i)];
			img[i][j] = (s >> 13 | s << 3);
		}
	}
#endif

	img.write(fused_full_name.c_str());

	string pose_txt_name = data_dir + "poseTSDF.txt";
	ofstream pose_file;
	pose_file.open(pose_txt_name.c_str(), fstream::app);
	pose_file.precision(60);

	for (int i = 0; i < 3; ++i) {
		pose_file << second_pose.data[i].x << "\t";
		pose_file << second_pose.data[i].y << "\t";
		pose_file << second_pose.data[i].z << "\t";
		pose_file << second_pose.data[i].w << "\n";
	}

	pose_file.close();
}

////////////////////////////////////////////////////////////////////////////////

bool GetExtrinsicData(string file_name, vector<Matrix4> *poses) {
	FILE *fp = fopen(file_name.c_str(), "r");
	for (int i = 0; i < image_list.size(); ++i) {
		Matrix4 m;
		for (int d = 0; d < 3; ++d) {
			if (fscanf(fp, "%f", &m.data[d].x));
			if (fscanf(fp, "%f", &m.data[d].y));
			if (fscanf(fp, "%f", &m.data[d].z));
			if (fscanf(fp, "%f", &m.data[d].w));
		}
		m.data[3].x = m.data[3].y = m.data[3].z = 0.f;
		m.data[3].w = 1.f;
		poses->push_back(m);
	}

	return true;
}

////////////////////////////////////////////////////////////////////////////////

bool GetImageData(string file_name, unsigned char *data) {
  unsigned char *raw_image = NULL;

  struct jpeg_decompress_struct cinfo;
  struct jpeg_error_mgr jerr;
  JSAMPROW row_pointer[1];

  FILE *infile = fopen(file_name.c_str(), "rb");
  unsigned long location = 0;

  if (!infile) {
    printf("Error opening jpeg file %s\n!", file_name.c_str());
    return -1;
  }
  cinfo.err = jpeg_std_error(&jerr);
  jpeg_create_decompress(&cinfo);
  jpeg_stdio_src(&cinfo, infile);
  jpeg_read_header(&cinfo, TRUE);
  jpeg_start_decompress(&cinfo);

  raw_image = (unsigned char*) malloc(
      cinfo.output_width * cinfo.output_height * cinfo.num_components);
  row_pointer[0] = (unsigned char *) malloc(
      cinfo.output_width * cinfo.num_components);
  while (cinfo.output_scanline < cinfo.image_height) {
    jpeg_read_scanlines(&cinfo, row_pointer, 1);
    for (uint i = 0; i < cinfo.image_width * cinfo.num_components; i++)
      raw_image[location++] = row_pointer[0][i];
  }

  int index = 0;
  for (uint i = 0; i < cinfo.image_height; ++i) {
    for (uint j = 0; j < cinfo.image_width; ++j) {
      for (int k = 0; k < kImageChannels; ++k) {
        *(data + index) = raw_image[(i * cinfo.image_width * 3) + (j * 3) + k];
        ++index;
      }
    }
  }

  jpeg_finish_decompress(&cinfo);
  jpeg_destroy_decompress(&cinfo);
  free(row_pointer[0]);
  fclose(infile);

  return true;
}

////////////////////////////////////////////////////////////////////////////////

int GetTimeStamp(const string &file_name) {
  return atoi(file_name.substr(
              file_name.size() - param_file_name_length + param_time_stamp_pose,
              param_time_stamp_length).c_str());
}

////////////////////////////////////////////////////////////////////////////////

void AssignDepthList(vector<string> image_list, vector<string> *depth_list) {
  vector<string> depth_temp;
  depth_temp.swap(*depth_list);
  depth_list->clear();
  depth_list->reserve(image_list.size());

  int idx = 0;
  int depth_time = GetTimeStamp(depth_temp[idx]);
  int time_low = depth_time;


  for (unsigned int i = 0; i < image_list.size(); ++i) {
    int image_time = GetTimeStamp(image_list[i]);

    while (depth_time < image_time) {
      if (idx == depth_temp.size() - 1)
        break;

      time_low = depth_time;
      depth_time = GetTimeStamp(depth_temp[++idx]);
    }

    if (idx == 0 && depth_time > image_time) {
      depth_list->push_back(depth_temp[idx]);
      continue;
    }

    if (abs(image_time - time_low) < abs(depth_time - image_time)) {
      depth_list->push_back(depth_temp[idx-1]);
    } else {
      depth_list->push_back(depth_temp[idx]);
    }
  }
}

////////////////////////////////////////////////////////////////////////////////

void SystemCommand(const string str) {
  if (system(str.c_str()))
    return;
}

////////////////////////////////////////////////////////////////////////////////

void ReComputeSecondPose() {
	if (param_start_index != depth_list.size() - 1) {
//		kfusion.ResetWeight(0.f);
//		GetDepthData(depth_list[param_start_index], (uint16_t *)depthImage.data());
//		kfusion.setKinectDeviceDepth(depthImage.getDeviceImage());
//		kfusion.setPose(toMatrix4(initPose));
//		kfusion.Integrate();
//		kfusion.Raycast();
//		hipDeviceSynchronize();

		Matrix4 delta = inverse(extrinsic_poses[param_start_index]) *
				                    extrinsic_poses[param_start_index + 1];
		kfusion.pose = kfusion.pose * delta;

		GetDepthData(depth_list[param_start_index + 1],
				         (uint16_t *)depthImage.data());
		kfusion.setKinectDeviceDepth(depthImage.getDeviceImage());
		hipDeviceSynchronize();

		kfusion.Track();
		hipDeviceSynchronize();

		second_pose = inverse(toMatrix4(initPose)) * kfusion.pose;
	}
}

////////////////////////////////////////////////////////////////////////////////

void display(void){
  static bool first_frame = true;
	static bool integrate = true;

    if (param_mode == KINFU_FORWARD) {
    	if (file_index == param_start_index + param_frame_threshold ||
    			file_index == image_list.size()) {
            param_mode = KINFU_BACKWARD;
            file_index = param_start_index - 1;
            kfusion.setPose(toMatrix4(initPose));

            kfusion.Raycast();
        		hipDeviceSynchronize();

    		cout << "IDX" << endl << endl;
            return;
    	}

#ifdef INITIAL_POSE
    	// T_12 = T_01^(-1) * T_02
    	// T_02 = T_01 * T_12;
    	if (file_index > 0 && file_index != param_start_index) {
    		Matrix4 delta = inverse(extrinsic_poses[file_index - 1]) *
    				            extrinsic_poses[file_index];
    		kfusion.pose = kfusion.pose * delta;
    	}
#endif
    } else {
    	if (file_index == param_start_index - param_frame_threshold ||
    			file_index == -1) {
    		kfusion.setPose(toMatrix4(initPose));
    		kfusion.Raycast();
    		hipDeviceSynchronize();

    		ReComputeSecondPose();

    		kfusion.setPose(toMatrix4(initPose));
    		kfusion.Raycast();
    		hipDeviceSynchronize();

    		SaveFusedDepthFile();

        cout << "IDX" << endl << endl;
    		exit(0);
    	}

#ifdef INITIAL_POSE
		Matrix4 delta = inverse(extrinsic_poses[file_index + 1]) *
				            extrinsic_poses[file_index];
		kfusion.pose = kfusion.pose * delta;
#endif
    }

    cout << file_index << " ";
    cout.flush();

    GetDepthData(depth_list[file_index], (uint16_t *)depthImage.data());
    kfusion.setKinectDeviceDepth(depthImage.getDeviceImage());

/*----------------------------------------------------------------------------*/

#if 0
    // Just integrate and raycast first frame
    kfusion.Integrate();
    kfusion.Raycast();
    SaveFusedDepthFile();
    exit(0);
#endif


#if 0
    // ICP off - actually on for integrate switch
    // extrinsic on
    Matrix4 temp = kfusion.pose;

    integrate = kfusion.Track();

    kfusion.pose = temp;

#else
    // ICP on
    integrate = kfusion.Track();

#endif

    double z_angle;
    Vector<3, float> diff_t;
    diff_t[0] = diff_t[1] = diff_t[2] = 0.f;

    if (file_index != param_start_index) {
			float3 cam_z;
			cam_z.x = cam_z.y = 0.f;
			cam_z.z = 1.f;
			float3 wor_z = kfusion.pose * cam_z;
			z_angle = acos(wor_z.z);

			float3 temp_t = kfusion.pose.get_translation();
			Vector<3, float> curr_t;
			curr_t[0] = temp_t.x;
			curr_t[1] = temp_t.y;
			curr_t[2] = temp_t.z;
			Vector<3, float> init_t = initPose.get_translation();
			diff_t = curr_t - init_t;
    }

    if ((!integrate && file_index != param_start_index) ||
//    		file_index == param_start_index + 14 ||
//    		file_index == param_start_index - 14 ||
    		z_angle > angle_threshold * param_angle_factor ||
    		norm(diff_t) > translation_threshold * param_translation_factor ) {
    	if (param_mode == KINFU_FORWARD) {
				param_mode = KINFU_BACKWARD;
				file_index = param_start_index - 1;
				kfusion.setPose(toMatrix4(initPose));

				kfusion.Raycast();
				hipDeviceSynchronize();

				cout << "THR" << endl << endl;
				return;
			} else {
				kfusion.setPose(toMatrix4(initPose));
				kfusion.Raycast();
				hipDeviceSynchronize();

    		ReComputeSecondPose();

    		kfusion.setPose(toMatrix4(initPose));
    		kfusion.Raycast();
    		hipDeviceSynchronize();

				SaveFusedDepthFile();

				cout << "THR" << endl << endl;

#if 0
// volume saving
				string vol_fn = fused_dir + "volume.txt";
				FILE *fpv = fopen(vol_fn.c_str(), "w");

				uint vol_size = kfusion.integration.size.x *
						            kfusion.integration.size.y *
						            kfusion.integration.size.z * sizeof(short2);

				short2 *vol_data = (short2*) malloc(vol_size);
				hipMemcpy(vol_data, kfusion.integration.data, vol_size,
						       hipMemcpyDeviceToHost);

				for (uint x = 0; x < kfusion.integration.size.x; ++x) {
					cout << x << endl;
					for (uint y = 0; y < kfusion.integration.size.y; ++y) {
						for (uint z = 0; z < kfusion.integration.size.z; ++z) {
							short2 data = vol_data[x +
							    y * kfusion.integration.size.x +
							    z * kfusion.integration.size.x * kfusion.integration.size.y];
							float2 dw = make_float2(data.x * 0.00003051944088f, data.y);
							fprintf(fpv, "%f %f ", dw.x, dw.y);
						}
					}
				}

				fclose(fpv);
#endif
				exit(0);
			}
    }

    if (param_mode == KINFU_FORWARD)
    	++file_index;
    else
    	--file_index;
/*----------------------------------------------------------------------------*/

    if(integrate || first_frame) {
        kfusion.Integrate();
        kfusion.Raycast();

        first_frame = false;
    }

    hipDeviceSynchronize();


    if(printCUDAError())
        exit(1);

//    usleep(1000 * 500);
}

////////////////////////////////////////////////////////////////////////////////

int main(int argc, char ** argv) {

	cout << "=================================================================" << endl;

	string server_prefix, data_prefix, server_dir, data_name;

	if (argc < 5) {
		cout << "Wrong arguments ..." << endl;
		exit(0);
	} else {
		server_prefix = argv[1];
		data_prefix = argv[2];
		data_name   = argv[3];
		param_start_index = atoi(argv[4]);
	}

	if (argc > 5)
		param_frame_threshold = atoi(argv[5]);
	if (argc > 6)
		param_volume_size = atoi(argv[6]);
	if (argc > 7)
		param_volume_dimension = atof(argv[7]);
	if (argc > 8)
		param_angle_factor = atof(argv[8]);
	if (argc > 9)
		param_translation_factor = atof(argv[9]);
	if (argc > 10)
		param_rsme_threshold = atof(argv[10]);

	server_dir = server_prefix + data_name;
	image_dir = server_dir + "image/";
	depth_dir = server_dir + "depth/";
	extrinsic_dir = server_dir + "extrinsics/";

	data_dir = data_prefix + data_name;

#ifdef RESOLUTION_1280X960
	fused_dir = data_dir + "depth1280x960/";
#else
	fused_dir = data_dir + "depthTSDF/";
#endif

  SystemCommand("mkdir -p " + fused_dir);

	file_index = param_start_index;

    size = param_volume_dimension;

    GetFileNames(image_dir, &image_list);
    GetFileNames(depth_dir, &depth_list);
    GetFileNames(extrinsic_dir, &extrinsic_list);
    AssignDepthList(image_list, &depth_list);

#ifdef INITIAL_POSE
    string extrinsic_name = extrinsic_list[extrinsic_list.size() - 1];
//    string extrinsic_name = extrinsic_list[1];

    GetExtrinsicData(extrinsic_name, &extrinsic_poses);
    cout << extrinsic_name << endl;
#endif

    float fx, fy, cx, cy, ff;
	string intrinsic = server_dir + "intrinsics.txt";
    FILE *fp = fopen(intrinsic.c_str(), "r");
    if (fscanf(fp, "%f", &fx));
    if (fscanf(fp, "%f", &ff));
    if (fscanf(fp, "%f", &cx));
    if (fscanf(fp, "%f", &ff));
    if (fscanf(fp, "%f", &fy));
    if (fscanf(fp, "%f", &cy));

    angle_threshold = (float) atan(cy / fy);
    translation_threshold = 1.0f * cy / fy;

/*----------------------------------------------------------------------------*/

    KFusionConfig config;

    config.volumeSize = make_uint3(param_volume_size);

    // these are physical dimensions in meters
    config.volumeDimensions = make_float3(size);
    config.nearPlane = 0.4f;
    config.farPlane = 5.0f;
    config.mu = 0.1;
    config.combinedTrackAndReduce = false;

    uint2 input_size = make_uint2(kImageCols, kImageRows);
    config.inputSize = input_size;

    config.camera = make_float4(fx, fy, cx, cy);

    config.rsme_threshold = param_rsme_threshold;

    config.iterations[0] = 10;
    config.iterations[1] = 5;
    config.iterations[2] = 4;

    initPose = SE3<float>(makeVector(size/2, size/2, 0, 0, 0, 0));

    kfusion.Init(config);

    // input buffers
    depthImage.alloc(input_size);

    // render buffers

    if(printCUDAError()) {
        hipDeviceReset();
        return 1;
    }

    memset(depthImage.data(), 0, depthImage.size.x * depthImage.size.y * sizeof(uint16_t));

#ifdef RESOLUTION_1280X960
    fusedDepth.alloc(input_size * 2);
#else
    fusedDepth.alloc(input_size);
#endif

    kfusion.setPose(toMatrix4(initPose));

    while(1) {
    	display();

    	if(stop_run)
    		break;
    }

    return 0;
}

////////////////////////////////////////////////////////////////////////////////

// sh run_sh ~/data/sun3d/ ~/data/sun3d/ hotel_umd/maryland_hotel3/
// scp maryland_hotel3.tar.gz alan@172.17.0.69:/home/alan/data/sun3d/hotel_umd/
